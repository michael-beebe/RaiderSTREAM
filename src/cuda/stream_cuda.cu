#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------*/
/* Program: RaiderSTREAM                                                 */
/* Original STREAM code developed by John D. McCalpin                    */
/* Programmers: Michael Beebe                                            */
/*              Brody Williams                                           */
/*              Pedro DaSilva                                            */
/*              Stephen Devaney                                          */
/* This program measures memory transfer rates in MB/s for simple        */
/* computational kernels coded in C.                                     */
/*-----------------------------------------------------------------------*/
/* License:                                                              */
/*  1. You are free to use this program and/or to redistribute           */
/*     this program.                                                     */
/*  2. You are free to modify this program for your own use,             */
/*     including commercial use, subject to the publication              */
/*     restrictions in item 3.                                           */
/*  3. Use of this program or creation of derived works based on this    */
/*     program constitutes acceptance of these licensing restrictions.   */
/*  4. Absolutely no warranty is expressed or implied.                   */
/*-----------------------------------------------------------------------*/

#include "stream_cuda_output.cuh"
#include "stream_cuda_tuned.cuh"
#include "stream_cuda_validation.cuh"
// #include "stream_cuda_kernels.cuh"

using namespace std;

// /*--------------------------------------------------------------------------------------
// - Initialize the STREAM arrays used in the kernels
// - Some compilers require an extra keyword to recognize the "restrict" qualifier.
// --------------------------------------------------------------------------------------*/
STREAM_TYPE* __restrict__   a;
STREAM_TYPE* __restrict__   b;
STREAM_TYPE* __restrict__   c;
STREAM_TYPE* __restrict__ d_a[device_count];
STREAM_TYPE* __restrict__ d_b[device_count];
STREAM_TYPE* __restrict__ d_c[device_count];

/*--------------------------------------------------------------------------------------
- Initialize IDX arrays (which will be used by gather/scatter kernels)
--------------------------------------------------------------------------------------*/
static ssize_t*   IDX1;
static ssize_t*   IDX2;
static ssize_t*   IDX3;
static ssize_t* d_IDX1[device_count];
static ssize_t* d_IDX2[device_count];
static ssize_t* d_IDX3[device_count];

/*--------------------------------------------------------------------------------------
- Initialize arrays to store avgtime, maxime, and mintime metrics for each kernel.
- The default values are 0 for avgtime and maxtime.
- each mintime[] value needs to be set to FLT_MAX via a for loop inside main()
--------------------------------------------------------------------------------------*/
static double avgtime[NUM_KERNELS] = {0};
static double maxtime[NUM_KERNELS] = {0};
static double mintime[NUM_KERNELS];
static int is_validated[NUM_KERNELS] = {0};

/*--------------------------------------------------------------------------------------
- Function to populate the STREAM arrays
--------------------------------------------------------------------------------------*/
void init_arrays(ssize_t array_elements) {
	ssize_t j;
	
	#pragma omp parallel for private (j)
    for (j = 0; j < array_elements; j++) {
		a[j] = 2.0;
		b[j] = 2.0;
		c[j] = 0.0;
    }
}

__global__ void stream_copy(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t array_elements) {
	ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[j] = d_a[j];
}

__global__ void stream_scale(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_b[j] = scalar * d_c[j];
}

__global__ void stream_sum(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[j] = d_a[j] + d_b[j];
}

__global__ void stream_triad(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_a[j] = d_b[j] + scalar * d_c[j];
}

void calculateTime(double t0, double times[NUM_KERNELS][NTIMES], int round, Kernels kernel) {
	hipDeviceSynchronize();
	times[kernel][round] = mysecond() - t0;
}

void executeSTREAM(STREAM_TYPE* __restrict__   a, STREAM_TYPE* __restrict__   b, STREAM_TYPE* __restrict__  c,
				   STREAM_TYPE* __restrict__ d_a[device_count], STREAM_TYPE* __restrict__ d_b[device_count], STREAM_TYPE* __restrict__ d_c[device_count],
				   ssize_t* __restrict__  d_IDX1[device_count], ssize_t* __restrict__  d_IDX2[device_count], ssize_t* __restrict__  d_IDX3[device_count],
				   double times[NUM_KERNELS][NTIMES], ssize_t stream_array_size, ssize_t array_elements, STREAM_TYPE scalar, int is_validated[NUM_KERNELS])
{
	init_arrays(array_elements);
	double t0;

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_a[deviceId], a, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_b[deviceId], b, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_c[deviceId], c, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
	}

	for(auto k = 0; k < NTIMES; k++) {
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			stream_copy<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], array_elements);
		}
		calculateTime(t0, times, k, COPY);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			stream_scale<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar, array_elements);
		}
		calculateTime(times, k, SCALE);

		
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			stream_sum<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar, array_elements);
		}
		calculateTime(times, k, SUM);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			stream_triad<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar, array_elements);
		}
		calculateTime(times, k, TRIAD);
	}

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(a + deviceId * array_elements, d_a[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(b + deviceId * array_elements, d_b[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(c + deviceId * array_elements, d_c[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
	}

	stream_validation(array_elements, scalar, is_validated, a, b, c);
}

__global__ void gather_copy(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[j] = d_a[d_IDX1[j]];
}

__global__ void gather_scale(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_b[j] = scalar * d_c[d_IDX2[j]];
}

__global__ void gather_sum(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[j] = d_a[d_IDX1[j]] + d_b[d_IDX2[j]];
}

__global__ void gather_triad(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_a[j] = d_b[d_IDX1[j]] + scalar * d_c[d_IDX2[j]];
}

void executeGATHER(STREAM_TYPE* __restrict__   a, STREAM_TYPE* __restrict__   b, STREAM_TYPE* __restrict__  c,
				   STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c,
				   ssize_t* __restrict__  d_IDX1[device_count], ssize_t* __restrict__  d_IDX2[device_count], ssize_t* __restrict__  d_IDX3[device_count],
				   double times[NUM_KERNELS][NTIMES], ssize_t stream_array_size, ssize_t array_elements, STREAM_TYPE scalar, int is_validated[NUM_KERNELS])
{
	init_arrays(array_elements);
	double t0;

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_a[deviceId], a, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_b[deviceId], b, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_c[deviceId], c, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
	}

	for(auto k = 0; k < NTIMES; k++) {
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			gather_copy<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, GATHER_COPY);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			gather_scale<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, GATHER_SCALE);

		
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			gather_sum<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, GATHER_SUM);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			gather_triad<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, GATHER_TRIAD);
	}

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(a + deviceId * array_elements, d_a[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(b + deviceId * array_elements, d_b[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(c + deviceId * array_elements, d_c[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
	}

	gather_validation(array_elements, scalar, is_validated, a, b, c);
}

__global__ void scatter_copy(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[d_IDX1[j]] = d_a[j];
}

__global__ void scatter_scale(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_b[d_IDX2[j]] = scalar * d_c[j];
}

__global__ void scatter_sum(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[d_IDX1[j]] = d_a[j] + d_b[j];
}

__global__ void scatter_triad(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_a[d_IDX2[j]] = d_b[j] + scalar * d_c[j];
}

void executeSCATTER(STREAM_TYPE* __restrict__   a, STREAM_TYPE* __restrict__   b, STREAM_TYPE* __restrict__  c,
				    STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c,
				    ssize_t* __restrict__  d_IDX1[device_count], ssize_t* __restrict__  d_IDX2[device_count], ssize_t* __restrict__  d_IDX3[device_count],
				    double times[NUM_KERNELS][NTIMES], ssize_t stream_array_size, ssize_t array_elements, STREAM_TYPE scalar, int is_validated[NUM_KERNELS])
{
	init_arrays(array_elements);
	double t0;

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_a[deviceId], a, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_b[deviceId], b, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_c[deviceId], c, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
	}

	for(auto k = 0; k < NTIMES; k++) {
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			scatter_copy<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SCATTER_COPY);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			scatter_scale<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SCATTER_SCALE);

		
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			scatter_sum<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SCATTER_SUM);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			scatter_triad<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SCATTER_TRIAD);
	}

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(a + deviceId * array_elements, d_a[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(b + deviceId * array_elements, d_b[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(c + deviceId * array_elements, d_c[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
	}

	scatter_validation(array_elements, scalar, is_validated, a, b, c);
}

__global__ void sg_copy(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t* __restrict__ d_IDX3, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[d_IDX1[j]] = d_a[d_IDX2[j]];
}

__global__ void sg_scale(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t* __restrict__ d_IDX3, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_b[d_IDX2[j]] = scalar * d_c[d_IDX1[j]];
}

__global__ void sg_sum(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t* __restrict__ d_IDX3, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_c[d_IDX1[j]] = d_a[d_IDX2[j]] + d_b[d_IDX3[j]];
}

__global__ void sg_triad(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t* __restrict__ d_IDX1, ssize_t* __restrict__ d_IDX2, ssize_t* __restrict__ d_IDX3, ssize_t array_elements) {
    ssize_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j < array_elements) d_a[d_IDX2[j]] = d_b[d_IDX3[j]] + scalar * d_c[d_IDX1[j]];
}

void executeSG(STREAM_TYPE* __restrict__   a, STREAM_TYPE* __restrict__   b, STREAM_TYPE* __restrict__  c,
			   STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c,
			   ssize_t* __restrict__  d_IDX1[device_count], ssize_t* __restrict__  d_IDX2[device_count], ssize_t* __restrict__  d_IDX3[device_count],
			   double times[NUM_KERNELS][NTIMES], ssize_t stream_array_size, ssize_t array_elements, STREAM_TYPE scalar, int is_validated[NUM_KERNELS])
{
	init_arrays(array_elements);
	double t0;

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_a[deviceId], a, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_b[deviceId], b, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_c[deviceId], c, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
	}

	for(auto k = 0; k < NTIMES; k++) {
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			sg_copy<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SG_COPY);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			sg_scale<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SG_SCALE);

		
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			sg_sum<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId],
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SG_SUM);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			sg_triad<<< (array_elements + 255)/256, 256 >>>(d_a[deviceId], d_b[deviceId], d_c[deviceId], scalar,
							d_IDX1[deviceId], d_IDX2[deviceId], d_IDX3[deviceId], array_elements);
		}
		calculateTime(t0, times, k, SG_TRIAD);
	}

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(a + deviceId * array_elements, d_a[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(b + deviceId * array_elements, d_b[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(c + deviceId * array_elements, d_c[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
	}

	sg_validation(array_elements, scalar, is_validated, a, b, c);
}

__global__ void central_copy(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t array_elements) {
	d_c[0] = d_a[0];
}

__global__ void central_scale(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t array_elements) {
	d_b[0] = scalar * d_c[0];
}

__global__ void central_sum(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, ssize_t array_elements) {
	d_c[0] = d_a[0] + d_b[0];
}

__global__ void central_triad(STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c, STREAM_TYPE scalar, ssize_t array_elements) {
	d_a[0] = d_b[0] + scalar * d_c[0];
}

void executeCENTRAL(STREAM_TYPE* __restrict__   a, STREAM_TYPE* __restrict__   b, STREAM_TYPE* __restrict__  c,
				   STREAM_TYPE* __restrict__ d_a, STREAM_TYPE* __restrict__ d_b, STREAM_TYPE* __restrict__ d_c,
				   ssize_t* __restrict__  d_IDX1[device_count], ssize_t* __restrict__  d_IDX2[device_count], ssize_t* __restrict__  d_IDX3[device_count],
				   double times[NUM_KERNELS][NTIMES], ssize_t stream_array_size, ssize_t array_elements, STREAM_TYPE scalar, int is_validated[NUM_KERNELS])
{
	init_arrays(array_elements);
	double t0;

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_a[deviceId], a, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_b[deviceId], b, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_c[deviceId], c, sizeof(STREAM_TYPE) * array_elements, hipMemcpyHostToDevice);
	}

	for(auto k = 0; k < NTIMES; k++) {
		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			central_copy<<< (array_elements + 255)/256, 256 >>>(d_a, d_b, d_c, array_elements);
		}
		calculateTime(t0, times, k, CENTRAL_COPY);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			central_scale<<< (array_elements + 255)/256, 256 >>>(d_a, d_b, d_c, scalar, array_elements);
		}
		calculateTime(t0, times, k, CENTRAL_SCALE);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			central_sum<<< (array_elements + 255)/256, 256 >>>(d_a, d_b, d_c, array_elements);
		}
		calculateTime(t0, times, k, CENTRAL_SUM);

		t0 = mysecond();
		for(auto deviceId = 0; deviceId < device_count; deviceId++) {
			hipSetDevice(deviceId);
			central_triad<<< (array_elements + 255)/256, 256 >>>(d_a, d_b, d_c, scalar, array_elements);
		}
		calculateTime(t0, times, k, CENTRAL_TRIAD);
	}

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(a + deviceId * array_elements, d_a[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(b + deviceId * array_elements, d_b[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
		hipMemcpy(c + deviceId * array_elements, d_c[deviceId], sizeof(STREAM_TYPE) * array_elements, hipMemcpyDeviceToHost);
	}

	central_validation(array_elements, scalar, is_validated, a, b, c);
}

#ifdef _OPENMP
extern int omp_get_num_threads();
#endif

int main(int argc, char *argv[]) {
    ssize_t stream_array_size = 10000000; // Default stream_array_size is 10000000
    int			quantum, checktick();
    ssize_t		j;
	double		t, times[NUM_KERNELS][NTIMES];
    STREAM_TYPE		scalar = 3.0;
	double		t0,t1,tmin;

/*
    get stream_array_size at runtime
*/
    parse_opts(argc, argv, &stream_array_size);

/*
    Allocate the arrays on the host
*/
	ssize_t array_elements = stream_array_size / device_count;
    a = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size);
    b = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size);
    c = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size);

	IDX1 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size);
	IDX2 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size);
    IDX3 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size);

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMalloc((void **) &d_a[deviceId], sizeof(STREAM_TYPE) * array_elements);
		hipMalloc((void **) &d_b[deviceId], sizeof(STREAM_TYPE) * array_elements);
		hipMalloc((void **) &d_c[deviceId], sizeof(STREAM_TYPE) * array_elements);

		hipMalloc((void **) &d_IDX1, sizeof(ssize_t) * array_elements);
		hipMalloc((void **) &d_IDX2, sizeof(ssize_t) * array_elements);
		hipMalloc((void **) &d_IDX3, sizeof(ssize_t) * array_elements);
	}

	double	bytes[NUM_KERNELS] = {
		// Original Kernels
		(double) 2 * sizeof(STREAM_TYPE) * stream_array_size, // Copy
		(double) 2 * sizeof(STREAM_TYPE) * stream_array_size, // Scale
		(double) 3 * sizeof(STREAM_TYPE) * stream_array_size, // Add
		(double) 3 * sizeof(STREAM_TYPE) * stream_array_size, // Triad
		// Gather Kernels
		(double) (((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // GATHER copy
		(double) (((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // GATHER Scale
		(double) (((3 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // GATHER Add
		(double) (((3 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // GATHER Triad
		// Scatter Kernels
		(double) (((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER copy
		(double) (((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Scale
		(double) (((3 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Add
		(double) (((3 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Triad
		// Scatter-Gather Kernels
		(double) (((2 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // SG copy
		(double) (((2 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // SG Scale
		(double) (((3 * sizeof(STREAM_TYPE)) + (3 * sizeof(ssize_t))) * stream_array_size), // SG Add
		(double) (((3 * sizeof(STREAM_TYPE)) + (3 * sizeof(ssize_t))) * stream_array_size), // SG Triad
		// Central Kernels
		(double) 2 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Copy
		(double) 2 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Scale
		(double) 3 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Add
		(double) 3 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Triad
	};

	double   flops[NUM_KERNELS] = {
		// Original Kernels
		(double) 0,                // Copy
		(double) 1 * stream_array_size, // Scale
		(double) 1 * stream_array_size, // Add
		(double) 2 * stream_array_size, // Triad
		// Gather Kernels
		(double) 0,                // GATHER Copy
		(double) 1 * stream_array_size, // GATHER Scale
		(double) 1 * stream_array_size, // GATHER Add
		(double) 2 * stream_array_size, // GATHER Triad
		// Scatter Kernels
		(double) 0,                // SCATTER Copy
		(double) 1 * stream_array_size, // SCATTER Scale
		(double) 1 * stream_array_size, // SCATTER Add
		(double) 2 * stream_array_size, // SCATTER Triad
        // Scatter-Gather Kernels
        (double) 0,
		(double) 1 * stream_array_size, // SCATTER Scale
		(double) 1 * stream_array_size, // SCATTER Add
		(double) 2 * stream_array_size, // SCATTER Triad
		// Central Kernels
		(double) 0,                // CENTRAL Copy
		(double) 1 * stream_array_size, // CENTRAL Scale
		(double) 1 * stream_array_size, // CENTRAL Add
		(double) 2 * stream_array_size, // CENTRAL Triad
	};

/*--------------------------------------------------------------------------------------
    - Set the mintime to default value (FLT_MAX) for each kernel, since we haven't executed
        any of the kernels or done any timing yet
--------------------------------------------------------------------------------------*/
    for (int i=0;i<NUM_KERNELS;i++) {
        mintime[i] = FLT_MAX;
    }

/*--------------------------------------------------------------------------------------
    - Initialize the idx arrays
	- Use the input .txt files to populate each array if the -DCUSTOM flag is enabled
	- If -DCUSTOM is not enabled, populate the IDX arrays with random values
--------------------------------------------------------------------------------------*/
#ifdef CUSTOM
	init_read_idx_array(IDX1, stream_array_size, "IDX1.txt");
	init_read_idx_array(IDX2, stream_array_size, "IDX2.txt");
	init_read_idx_array(IDX3, stream_array_size, "IDX2.txt");
#else
    srand(time(0));
    init_random_idx_array(IDX1, array_elements);
    init_random_idx_array(IDX2, array_elements);
    init_random_idx_array(IDX3, array_elements);
#endif

	for(auto deviceId = 0; deviceId < device_count; deviceId++) {
		hipSetDevice(deviceId);

		hipMemcpy(d_IDX1, IDX1, sizeof(ssize_t) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_IDX2, IDX2, sizeof(ssize_t) * array_elements, hipMemcpyHostToDevice);
		hipMemcpy(d_IDX3, IDX3, sizeof(ssize_t) * array_elements, hipMemcpyHostToDevice);
	}

/*--------------------------------------------------------------------------------------
    - Print initial info
--------------------------------------------------------------------------------------*/
    print_info1(stream_array_size);

#ifdef _OPENMP
    printf(HLINE);
#pragma omp parallel
    {
#pragma omp master
	   {
    	    k = omp_get_num_threads();
    	    printf ("Number of Threads requested = %i\n",k);
        }
    }
#endif
#ifdef _OPENMP
	k = 0;
#pragma omp parallel
#pragma omp atomic
		k++;
    printf ("Number of Threads counted = %i\n",k);
#endif

/*--------------------------------------------------------------------------------------
    // Populate STREAM arrays
--------------------------------------------------------------------------------------*/
#pragma omp parallel for private (j)
    for (j=0; j<stream_array_size; j++) {
        a[j] = 1.0;
        b[j] = 2.0;
        c[j] = 0.0;
    }

/*--------------------------------------------------------------------------------------
    // Estimate precision and granularity of timer
--------------------------------------------------------------------------------------*/
	print_timer_granularity(quantum);

    t = mysecond();
#pragma omp parallel for private (j)
    for (j = 0; j < stream_array_size; j++) {
  		a[j] = 2.0E0 * a[j];
	}

    t = 1.0E6 * (mysecond() - t);

	print_info2(t, quantum);
	print_memory_usage(stream_array_size);

	executeSTREAM( a, b, c, d_a, d_b, d_c, d_IDX1, d_IDX2, d_IDX3, times, stream_array_size, array_elements, scalar, is_validated);
	executeGATHER( a, b, c, d_a, d_b, d_c, d_IDX1, d_IDX2, d_IDX3, times, stream_array_size, array_elements, scalar, is_validated);
	executeSCATTER(a, b, c, d_a, d_b, d_c, d_IDX1, d_IDX2, d_IDX3, times, stream_array_size, array_elements, scalar, is_validated);
	executeSG(     a, b, c, d_a, d_b, d_c, d_IDX1, d_IDX2, d_IDX3, times, stream_array_size, array_elements, scalar, is_validated);
	executeCENTRAL(a, b, c, d_a, d_b, d_c, d_IDX1, d_IDX2, d_IDX3, times, stream_array_size, array_elements, scalar, is_validated);

/*--------------------------------------------------------------------------------------
	// Calculate results
--------------------------------------------------------------------------------------*/
    for (int k=1; k<NTIMES; k++) /* note -- skip first iteration */
	{
	for (j=0; j<NUM_KERNELS; j++)
	    {
			avgtime[j] = avgtime[j] + times[j][k];
			mintime[j] = MIN(mintime[j], times[j][k]);
			maxtime[j] = MAX(maxtime[j], times[j][k]);
	    }
	}

/*--------------------------------------------------------------------------------------
	// Print results table
--------------------------------------------------------------------------------------*/
    printf("Function\tBest Rate MB/s      Best FLOP/s\t   Avg time\t   Min time\t   Max time\n");
    for (j=0; j<NUM_KERNELS; j++) {
		avgtime[j] = avgtime[j]/(double)(NTIMES-1);

		if (j % 4 == 0) {
			printf(HLINE);
		}

        if (flops[j] == 0) {
            printf("%s%12.1f\t\t%s\t%11.6f\t%11.6f\t%11.6f\n",
                label[j].c_str(),                           // Kernel
                1.0E-06 * bytes[j]/mintime[j],      // MB/s
                "-",      // FLOP/s
                avgtime[j],                         // Avg Time
                mintime[j],                         // Min Time
                maxtime[j]);                        // Max time
        }
        else {
            printf("%s%12.1f\t%12.1f\t%11.6f\t%11.6f\t%11.6f\n",
                label[j].c_str(),                           // Kernel
                1.0E-06 * bytes[j]/mintime[j],      // MB/s
                1.0E-06 * flops[j]/mintime[j],      // FLOP/s
                avgtime[j],                         // Avg Time
                mintime[j],                         // Min Time
                maxtime[j]);                        // Max time
        }
    }
    printf(HLINE);

/*--------------------------------------------------------------------------------------
	// Validate results
--------------------------------------------------------------------------------------*/
	checkSTREAMresults(is_validated);
    printf(HLINE);

	free(a);
	free(b);
	free(c);

	free(IDX1);
	free(IDX2);
	free(IDX3);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	hipFree(d_IDX1);
	hipFree(d_IDX2);
	hipFree(d_IDX3);

    return 0;
}