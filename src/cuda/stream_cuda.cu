#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------*/
/* Program: RaiderSTREAM                                                 */
/* Original STREAM code developed by John D. McCalpin                    */
/* Programmers: Michael Beebe                                            */
/*              Brody Williams                                           */
/*              Pedro DaSilva                                            */
/*              Stephen Devaney                                          */
/* This program measures memory transfer rates in MB/s for simple        */
/* computational kernels coded in C.                                     */
/*-----------------------------------------------------------------------*/
/* License:                                                              */
/*  1. You are free to use this program and/or to redistribute           */
/*     this program.                                                     */
/*  2. You are free to modify this program for your own use,             */
/*     including commercial use, subject to the publication              */
/*     restrictions in item 3.                                           */
/*  3. Use of this program or creation of derived works based on this    */
/*     program constitutes acceptance of these licensing restrictions.   */
/*  4. Absolutely no warranty is expressed or implied.                   */
/*-----------------------------------------------------------------------*/

#include "stream_cuda_output.cuh"
#include "stream_cuda_tuned.cuh"
#include "stream_cuda_validation.cuh"

// /*--------------------------------------------------------------------------------------
// - Initialize the STREAM arrays used in the kernels
// - Some compilers require an extra keyword to recognize the "restrict" qualifier.
// --------------------------------------------------------------------------------------*/
STREAM_TYPE * restrict a;
STREAM_TYPE * restrict b;
STREAM_TYPE * restrict c;

/*--------------------------------------------------------------------------------------
- Initialize IDX arrays (which will be used by gather/scatter kernels)
--------------------------------------------------------------------------------------*/
static ssize_t *IDX1;
static ssize_t *IDX2;
static ssize_t *IDX3;

/*--------------------------------------------------------------------------------------
- Initialize arrays to store avgtime, maxime, and mintime metrics for each kernel.
- The default values are 0 for avgtime and maxtime.
- each mintime[] value needs to be set to FLT_MAX via a for loop inside main()
--------------------------------------------------------------------------------------*/
static double avgtime[NUM_KERNELS] = {0};
static double maxtime[NUM_KERNELS] = {0};
static double mintime[NUM_KERNELS];
static int is_validated[NUM_KERNELS] = {0};

/*--------------------------------------------------------------------------------------
- Function to populate the STREAM arrays
--------------------------------------------------------------------------------------*/
void init_arrays(ssize_t stream_array_size) {
	ssize_t j;
	
	#pragma omp parallel for private (j)
    for (j = 0; j < stream_array_size; j++) {
		a[j] = 2.0; // 1 or 2? since we are changing the validation we could discuss
		b[j] = 2.0;
		c[j] = 0.0;
    }
}

//////////////////////////////////////////////////////////////////////////////

__global__ void stream_copy(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
    
}

__global__ void stream_scale(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void stream_sum(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void stream_triad(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}


__global__ void gather_copy(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void gather_scale(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void gather_sum(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void gather_triad(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}


__global__ void scatter_copy(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void scatter_scale(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void scatter_sum(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void scatter_triad(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}


__global__ void sg_copy(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void sg_scale(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void sg_sum(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void sg_triad(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}


__global__ void central_copy(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void central_scale(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void central_sum(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

__global__ void central_triad(ssize_t stream_array_size, double times[NUM_KERNELS][NTIMES], int k, STREAM_TYPE scalar) {
    //TODO:
}

//////////////////////////////////////////////////////////////////////////////

#ifdef _OPENMP
extern int omp_get_num_threads();
#endif

int main(int argc, char *argv[]) {
    ssize_t stream_array_size = 10000000; // Default stream_array_size is 10000000
    int			quantum, checktick();
    int			BytesPerWord;
    int			k;
    ssize_t		j;
    STREAM_TYPE		scalar;
    double		t, times[NUM_KERNELS][NTIMES];
	double		t0,t1,tmin;

/*
    get stream_array_size at runtime
*/
    parse_opts(argc, argv, &stream_array_size);

/*
    Allocate the arrays on the host
*/
    a = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size+OFFSET);
    b = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size+OFFSET);
    c = (STREAM_TYPE *) malloc(sizeof(STREAM_TYPE) * stream_array_size+OFFSET);

	IDX1 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size+OFFSET);
	IDX2 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size+OFFSET);
    IDX3 = (ssize_t *) malloc(sizeof(ssize_t) * stream_array_size+OFFSET);

	double	bytes[NUM_KERNELS] = {
		// Original Kernels
		2 * sizeof(STREAM_TYPE) * stream_array_size, // Copy
		2 * sizeof(STREAM_TYPE) * stream_array_size, // Scale
		3 * sizeof(STREAM_TYPE) * stream_array_size, // Add
		3 * sizeof(STREAM_TYPE) * stream_array_size, // Triad
		// Gather Kernels
		(((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // GATHER copy
		(((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // GATHER Scale
		(((3 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // GATHER Add
		(((3 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // GATHER Triad
		// Scatter Kernels
		(((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER copy
		(((2 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Scale
		(((3 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Add
		(((3 * sizeof(STREAM_TYPE)) + (1 * sizeof(ssize_t))) * stream_array_size), // SCATTER Triad
		// Scatter-Gather Kernels
		(((2 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // SG copy
		(((2 * sizeof(STREAM_TYPE)) + (2 * sizeof(ssize_t))) * stream_array_size), // SG Scale
		(((3 * sizeof(STREAM_TYPE)) + (3 * sizeof(ssize_t))) * stream_array_size), // SG Add
		(((3 * sizeof(STREAM_TYPE)) + (3 * sizeof(ssize_t))) * stream_array_size), // SG Triad
		// Central Kernels
		2 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Copy
		2 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Scale
		3 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Add
		3 * sizeof(STREAM_TYPE) * stream_array_size, // CENTRAL Triad
	};

	double   flops[NUM_KERNELS] = {
		// Original Kernels
		(int)0,                // Copy
		1 * stream_array_size, // Scale
		1 * stream_array_size, // Add
		2 * stream_array_size, // Triad
		// Gather Kernels
		(int)0,                // GATHER Copy
		1 * stream_array_size, // GATHER Scale
		1 * stream_array_size, // GATHER Add
		2 * stream_array_size, // GATHER Triad
		// Scatter Kernels
		(int)0,                // SCATTER Copy
		1 * stream_array_size, // SCATTER Scale
		1 * stream_array_size, // SCATTER Add
		2 * stream_array_size, // SCATTER Triad
        // Scatter-Gather Kernels
        (int)0,
		1 * stream_array_size, // SCATTER Scale
		1 * stream_array_size, // SCATTER Add
		2 * stream_array_size, // SCATTER Triad
		// Central Kernels
		(int)0,                // CENTRAL Copy
		1 * stream_array_size, // CENTRAL Scale
		1 * stream_array_size, // CENTRAL Add
		2 * stream_array_size, // CENTRAL Triad
	};

/*--------------------------------------------------------------------------------------
    - Set the mintime to default value (FLT_MAX) for each kernel, since we haven't executed
        any of the kernels or done any timing yet
--------------------------------------------------------------------------------------*/
    for (int i=0;i<NUM_KERNELS;i++) {
        mintime[i] = FLT_MAX;
    }

/*--------------------------------------------------------------------------------------
    - Initialize the idx arrays
	- Use the input .txt files to populate each array if the -DCUSTOM flag is enabled
	- If -DCUSTOM is not enabled, populate the IDX arrays with random values
--------------------------------------------------------------------------------------*/
#ifdef CUSTOM
	init_read_idx_array(IDX1, stream_array_size, "IDX1.txt");
	init_read_idx_array(IDX2, stream_array_size, "IDX2.txt");
	init_read_idx_array(IDX3, stream_array_size, "IDX2.txt");
#else
    srand(time(0));
    init_random_idx_array(IDX1, stream_array_size);
    init_random_idx_array(IDX2, stream_array_size);
    init_random_idx_array(IDX3, stream_array_size);
#endif

/*--------------------------------------------------------------------------------------
    - Print initial info
--------------------------------------------------------------------------------------*/
    print_info1(BytesPerWord, stream_array_size);

#ifdef _OPENMP
    printf(HLINE);
#pragma omp parallel
    {
#pragma omp master
	   {
    	    k = omp_get_num_threads();
    	    printf ("Number of Threads requested = %i\n",k);
        }
    }
#endif
#ifdef _OPENMP
	k = 0;
#pragma omp parallel
#pragma omp atomic
		k++;
    printf ("Number of Threads counted = %i\n",k);
#endif

/*--------------------------------------------------------------------------------------
    // Populate STREAM arrays
--------------------------------------------------------------------------------------*/
#pragma omp parallel for private (j)
    for (j=0; j<stream_array_size; j++) {
        a[j] = 1.0;
        b[j] = 2.0;
        c[j] = 0.0;
    }

// TODO: copy necessary data to device

/*--------------------------------------------------------------------------------------
    // Estimate precision and granularity of timer
--------------------------------------------------------------------------------------*/
	print_timer_granularity(quantum);

    t = mysecond();
#pragma omp parallel for private (j)
    for (j = 0; j < stream_array_size; j++) {
  		a[j] = 2.0E0 * a[j];
	}

    t = 1.0E6 * (mysecond() - t);

	print_info2(t, quantum);
	print_memory_usage(stream_array_size);

    scalar = 3.0;


// TODO: kernels

}